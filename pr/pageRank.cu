#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <mpi.h>
#include <time.h>

int* d_vertices;
int* d_edges;
int* d_out_degrees;
int* d_destinations;

//current pr values
double* d_x;
//new pr values
double* d_y;

double* d_weight;

int* d_num_vertices;
int* d_num_edges;
int* d_elementsPerProc;
int* d_done;
int* d_offset;

int world_rank;
int world_size;
int blocks;
int threads;
int offset;

void log(double* x, int num_vertices){
    int i;
    for(i=0; i < num_vertices; i++){
        printf("x[%d] = %1f\n", i, x[i]);
    }
}

__global__ void CUDA_INIT_PR_VALUES(double* d_x, int* d_num_vertices){
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    d_x[id] = 1.0f / *d_num_vertices;
    if(id < 10){
        double res = (double)1/2692096;
        printf("GPU float: %5f\n", d_x[id]);
        printf("GPU res: %.20f\n", res);
    }

}

//d_y is the output page rank
//d_x is the old values
//d_destinations is the edges
//d_vertices is the sources
__global__ void CUDA_ITERATE_KERNEL(int* d_vertices, int* d_destinations, double* d_x, double* d_y, int* d_out_degrees, int* d_num_vertices, int* d_offset){

    double d = 0.85;
    int i;
    int s;
    double sum = 0;

    //Set the thread id
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if(d_out_degrees[id + *d_offset ] != 0){
        for(i = d_vertices[id + *d_offset]; i < d_vertices[id +1 + *d_offset]; i++){
            s = d_destinations[i];

            if(d_out_degrees[s] != 0){
                // new result += previous values / number of out degrees
                sum += d * d_x[s] / d_out_degrees[s]; // Check this
            }
        }
        //Likely need to add this outside of kernel when using MPI. Talk with Hans as this is constant
        sum += (1 - d) / *d_num_vertices;



        d_y[id  + *d_offset] = sum;
    }
}
//Do ALL to all after this

__global__ void CUDA_WEIGHTS_KERNEL(double* d_y, double* d_weight, int* d_num_vertices){

    int id = threadIdx.x + blockIdx.x * blockDim.x;

    d_y[id] += *d_weight * (1.0f / *d_num_vertices);

}

__global__ void CUDA_SCALE_SWAP_KERNEL(double* d_x, double* d_y, double* d_weight){
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    d_x[id] = *d_weight * d_y[id];
    d_y[id] = 0;
}

void setup(int* vertices, int* destinations, int* out_degrees, int num_vertices, int num_edges, double* x, double* y, int offset,
            int numLocalVertices){
    hipMalloc((void**)&d_vertices, sizeof(int) * num_vertices);
    hipMemcpy(d_vertices, vertices, sizeof(int) * num_vertices, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_destinations, sizeof(int) * num_edges);
    hipMemcpy(d_destinations, destinations, sizeof(int) * num_edges, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_out_degrees, sizeof(int) * num_vertices);
    hipMemcpy(d_out_degrees, out_degrees, sizeof(int) * num_vertices, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_num_vertices, sizeof(int));
    hipMemcpy(d_num_vertices, &num_vertices, sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_num_edges, sizeof(int));
    hipMemcpy(d_num_edges, &num_edges, sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_x, sizeof(double) * num_vertices);
    hipMemcpy(d_x, x, sizeof(double) * num_vertices, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_y, sizeof(double) * num_vertices);

    hipMalloc((void**)&d_weight, sizeof(double));

    hipMalloc((void**)&d_offset, sizeof(int));
    hipMemcpy(d_offset, &offset, sizeof(int), hipMemcpyHostToDevice);

    CUDA_INIT_PR_VALUES <<<blocks, threads>>> (d_x, d_num_vertices);
    printf("Init PR values Complete\n");
    hipMemcpy(x, d_x, sizeof(double) * num_vertices, hipMemcpyDeviceToHost);
    hipMemcpy(y, d_x, sizeof(double) * num_vertices, hipMemcpyDeviceToHost);

    printf("Setup Complete\n");
}

void iterate(double* inX, double* outX, double* inY, double* outY, int num_vertices){

    hipMemcpy(d_x, inX, sizeof(double) * num_vertices, hipMemcpyHostToDevice);
    hipMemcpy(d_y, inY, sizeof(double) * num_vertices, hipMemcpyHostToDevice);

    CUDA_ITERATE_KERNEL <<<blocks, threads >>> (d_vertices, d_destinations, d_x, d_y, d_out_degrees, d_num_vertices, d_offset);

    hipMemcpy(outX, d_x, sizeof(double) * num_vertices, hipMemcpyDeviceToHost);
    hipMemcpy(outY, d_y, sizeof(double) * num_vertices, hipMemcpyDeviceToHost);

}

double sum(double* array, int length){
    double sum = 0;
    double err = 0;
    int i;
    for(i = 0; i < length; i++){
        double tmp = sum;
        double y = array[i] + err;
        sum = tmp + y;
        err = tmp - sum;
        err += y;
    }
    return sum;
}

//Calculate manhatten distance between input and output
double normdiff(double* input, double* output, int length){
    double d = 0;
    double err = 0;
    int i;
    for(i = 0; i < length; i++){
        double tmp = d;
        double y = abs(output[i] - input[i]) + err;
        d = tmp + y;
        err = tmp - d;
        err += y;
    }
    return d;
}

int factor(int length){
    int res = 1;
    int i;
    for(i=1; i <= length && i <= 1024; i++){
        if(length % i == 0){
            res = i;
        }
    }
    return res;
}



void pageRank(int* vertices, int num_vertices, int* destinations, int num_destinations, int* outDegrees, int* verticesStarts,
                int world_rank, int world_size, int num_edges){

    int numLocalVertices;
    int i;

    int maxIterations = 100;
    int iteration = 1;
    double tol = 0.0000005;
    double delta = 2;

    int localDispl = verticesStarts[world_rank];

    double* x = (double *) malloc( num_vertices * sizeof(double));
    double* y = (double *) malloc( num_vertices * sizeof(double));


    if(world_rank < world_size - 1){
        numLocalVertices = (verticesStarts[world_rank + 1] - verticesStarts[world_rank]);
    }
    else{
        numLocalVertices = (num_vertices - verticesStarts[world_rank]);
    }
    printf("numLocalVertices: %d\n", numLocalVertices);

    double startExecution = MPI_Wtime();

    int *recvcounts = NULL;
    if(world_rank == 0){
        recvcounts = (int *) malloc(world_size * sizeof(int));
    }

    MPI_Gather(&numLocalVertices, 1, MPI_INT, recvcounts, 1, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Barrier(MPI_COMM_WORLD);

    if(world_rank == 0){
        for(i = 0; i < world_size; i++){
            printf("recv count [%d] = %d\n", i, recvcounts[i]);
        }
    }


    threads = factor(numLocalVertices);
    blocks =  numLocalVertices / threads;


    printf("threads: %d\n", threads);
    printf("blocks: %d\n", blocks);

    int totlen = 0;
    int* displs = NULL;
    double* globalY = NULL;
    double* globalX = NULL;

    globalX = (double* ) malloc ( num_vertices * sizeof(double) );
    globalY = (double* ) malloc ( num_vertices * sizeof(double) );

    if(world_rank == 0){
        displs = (int* ) malloc ( world_size * sizeof(int) );
        displs[0] = 0;
        totlen = recvcounts[0];


        for(i=1 ; i < world_size; i++){
            totlen += recvcounts[i];
            displs[i] = displs[i-1] + recvcounts[i-1];
        }
    }

    setup(vertices, destinations, outDegrees, num_vertices, num_destinations, x, y, localDispl, numLocalVertices);


    MPI_Gatherv(x, numLocalVertices, MPI_DOUBLE, globalX, recvcounts, displs, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    MPI_Gatherv(y, numLocalVertices, MPI_DOUBLE, globalY, recvcounts, displs, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    MPI_Bcast(globalX, num_vertices, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    MPI_Bcast(globalY, num_vertices, MPI_DOUBLE, 0, MPI_COMM_WORLD);

    double startIterationTime;
    double endIterationTime;

    double startCommunicationTime;
    double endCommunicationTime;


    clock_t start;
    clock_t diff;
    int msec;
    double verticesPermSec;
    while(iteration < maxIterations && delta > tol){

        startIterationTime = MPI_Wtime();

        //Each process except master resets y and global Y to 0.
        //Better than doing broadcast at end of while loop to stop network bottleneck
        if(iteration > 0 && world_rank != 0){
            memset(y, 0, num_vertices * sizeof(double));
            memset(globalY, 0, num_vertices * sizeof(double));
        }

        //call iterations
        start = clock();
        iterate(globalX, x, globalY, y, num_vertices);
        diff = clock() - start;
        msec = diff * 1000 / CLOCKS_PER_SEC;
        printf("Process: %d -> Time taken %d seconds %d milliseconds\n", world_rank, msec/1000, msec%1000);
        printf("Process: %d -> Vertices processed per msec  %.5f\n", world_rank, (double)num_vertices/(msec%1000));
        printf("Process: %d -> edges processed per msec  %.5f\n", world_rank, (double)num_edges/(msec%1000));

        MPI_Reduce(y, globalY, num_vertices, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);

        if(world_rank == 0 && iteration > 1){
            for(i=0; i < 50; i++){
                //printf("globalY[%d]: %.50f\n", i, globalY[i]);
        }
        }

        if(world_rank == 0){
            double weight = 1.0f - sum(globalY, num_vertices); //ensure y[] sums to 1
            printf("weight: %.5f\n", weight);


        hipMemcpy(d_weight, &weight, sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_y, globalY, sizeof(double) * num_vertices, hipMemcpyHostToDevice);

        threads = factor(num_vertices);
        blocks = num_vertices / threads;

        CUDA_WEIGHTS_KERNEL<<<blocks, threads>>>(d_y, d_weight, d_num_vertices);

        hipMemcpy(globalY, d_y, sizeof(double) * num_vertices, hipMemcpyDeviceToHost);

        delta = normdiff(globalX, globalY, num_vertices);
        printf("Iteration: %d - Delta: %1f\n", iteration, delta);

        //rescale to unit length
        weight = 1.0f / sum(globalY, num_vertices);
        printf("After rescale: weight: %6f\n", weight);

        hipMemcpy(d_weight, &weight, sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_y, globalY, sizeof(double) * num_vertices, hipMemcpyHostToDevice);
        hipMemcpy(d_x, globalX, sizeof(double) * num_vertices, hipMemcpyHostToDevice);
        CUDA_SCALE_SWAP_KERNEL<<<blocks, threads>>>(d_x, d_y, d_weight);
        hipMemcpy(globalX, d_x, sizeof(double) * num_vertices, hipMemcpyDeviceToHost);
        hipMemcpy(y, d_y, sizeof(double) * num_vertices, hipMemcpyDeviceToHost);
        hipMemcpy(globalY, d_y, sizeof(double) * num_vertices, hipMemcpyDeviceToHost);

        }

        startCommunicationTime = MPI_Wtime();
        MPI_Bcast(&delta, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
        MPI_Bcast(globalX, num_vertices, MPI_DOUBLE, 0, MPI_COMM_WORLD);
        endCommunicationTime = MPI_Wtime();
        //MPI_Bcast(globalY, num_vertices, MPI_DOUBLE, 0, MPI_COMM_WORLD);
        //MPI_Bcast(y, num_vertices, MPI_DOUBLE, 0, MPI_COMM_WORLD);

        if(world_rank == 0){
            endIterationTime = MPI_Wtime();
            printf("Iteration: %d - Time taken: %.6f s\n", iteration, endIterationTime - startIterationTime);

            printf("Iteration: %d - Communication Time taken: %.6f s\n", iteration, endCommunicationTime - startCommunicationTime);

        }
        iteration++;
    }
    MPI_Barrier(MPI_COMM_WORLD);

    double finishExecution = MPI_Wtime();

    if(world_rank == 0){

    if(delta > tol){

        printf("\n");
        printf("Execution Time: %.6f\n", finishExecution - startExecution);
        printf("\n");
        printf("No convergence\n");

        int i;
        for(i = 0; i < 10; i++){
            printf("x[%d] = %.70f\n", i, globalX[i]);
        }
    }
    else{
        printf("\n");
        printf("Execution Time: %.6f s\n", finishExecution - startExecution);
        printf("\n");
        printf("Convergence at iteration %d \n", iteration - 1);
        printf("\n");
        printf("Values:\n");

        int i;
        for(i =0; i < num_vertices; i++){
            printf("x[%d] = %.70f\n", i, globalX[i]);
        }
    }
    }
}
